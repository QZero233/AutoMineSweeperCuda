#include "hip/hip_runtime.h"
#include "cudaWorker.cuh"
#include<algorithm>

__device__ int getIdx(int x, int y, Image* img) {
	return y * img->colNum + x;
}

//__global__ void freeImageCuda(Image* image) {
//	jint* ptr = image->ptr;
//	hipFree(ptr);
//	hipFree(image);
//}

__global__ void solveCuda(Image* game, jlong* targets, jdouble* result, int targetSize, Profile* profile) {

	int targetX = blockIdx.x;
	int targetY = blockIdx.y;

	if (targetX >= profile->xNum || targetY >= profile->yNum)
		return;

	int index = threadIdx.x;
	int stride = blockDim.x;
	for (int i = index; i < targetSize; i += stride){
		//Calculate target i for (x,y)
		Image* target = (Image*)targets[i];

		Coordinate leftTop;
		leftTop.x = profile->xDivides[targetX];
		leftTop.y = profile->yDivides[targetY];

		Coordinate rightBottom;
		rightBottom.x = profile->xDivides[targetX + 1];
		rightBottom.y = profile->yDivides[targetY + 1];

		int xSize = rightBottom.x - leftTop.x + 1;
		int ySize = rightBottom.y - leftTop.y + 1;

		int xMin = min(xSize, target->colNum);
		int yMin = min(ySize, target->rowNum);

		double currentSum = 0;
		for (int x = xMin / 8; x < xMin * 7 / 8; x++) {
			for (int y = yMin / 8; y < yMin * 7 / 8; y++) {
				int srcColor = game->ptr[getIdx(leftTop.x + x, leftTop.y + y, game)];
				int targetColor = target->ptr[getIdx(x, y, target)];

				currentSum += std::abs(targetColor - srcColor);
			}
		}

		int targetOffset = targetSize * (targetY * profile->xNum + targetX);
		result[targetOffset + i] = currentSum / (xMin * yMin);
	}
}

void solve(Image* game, jlong* targets, jdouble* result, int targetSize, Profile* profile) {
	dim3 blockSize((targetSize / 32 + 1) * 32);
	dim3 gridSize(128,128);

	//Copy targets array
	jlong* cudaTargets = NULL;
	hipMalloc((void**)&cudaTargets, targetSize * sizeof(jlong*));
	hipMemcpy(cudaTargets, targets, targetSize * sizeof(jlong*), hipMemcpyHostToDevice);

	solveCuda << <gridSize, blockSize >> > (game, cudaTargets, result, targetSize, profile);

	hipFree(cudaTargets);
}

//void freeImage(Image* image) {
//	dim3 blockSize(1);
//	dim3 gridSize(1);
//	//freeImageCuda<<<gridSize, blockSize>>>(image);
//}